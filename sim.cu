
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
#include<cmath>
#include<fstream>

#include<chrono>

#define SIM_STATE_NAME "/sim_state.bin"



//shared host/device constants
int gridWidth,gridHeight,gridDepth,blockWidth,blockHeight,blockDepth,gridWidthBlocks,gridHeightBlocks,gridDepthBlocks,gridArea;
__constant__ int gridWidth_d,gridHeight_d,gridDepth_d,blockWidth_d,blockHeight_d,blockDepth_d,gridWidthBlocks_d,gridHeightBlocks_d,gridDepthBlocks_d;

//host only constants
int timeSteps;



__device__ int arrayPos(const int &x,const int &y,const int &z){
    return (z*gridWidth_d*gridHeight_d)+(y*gridWidth_d)+x;
}



__global__ void solver(double *grid,double *grid1){
    int xpos = (blockIdx.x*blockWidth_d)+threadIdx.x;
    int ypos = (blockIdx.y*blockHeight_d)+threadIdx.y;
    int zpos = (blockIdx.z*blockDepth_d)+threadIdx.z;

    if(xpos>0 && xpos<gridWidth_d-1 && ypos>0 && ypos<gridHeight_d-1 && zpos>0 && zpos<gridDepth_d-1){
        //the grid1 value at each point contains the value from 2 steps ago(but is overwritten once used)
        grid1[arrayPos(xpos,ypos,zpos)] = grid[arrayPos(xpos+1,ypos,zpos)]+grid[arrayPos(xpos-1,ypos,zpos)]
                                        +grid[arrayPos(xpos,ypos+1,zpos)]+grid[arrayPos(xpos,ypos-1,zpos)]
                                        +grid[arrayPos(xpos,ypos,zpos+1)]+grid[arrayPos(xpos,ypos,zpos-1)]
                                        -grid1[arrayPos(xpos,ypos,zpos)];
    }
}



//helper function to read header from a binary file
void readHeaderBinary(FILE *fileIn,int *w,int *h,int *d){
    fread(w,sizeof(int),1,fileIn);
    fread(h,sizeof(int),1,fileIn);
    fread(d,sizeof(int),1,fileIn);
}

//helper function to write header to a binary file
void writeHeaderBinary(FILE *fileOut,int *w,int *h,int *d){
    fwrite(w,sizeof(int),1,fileOut);
    fwrite(h,sizeof(int),1,fileOut);
    fwrite(d,sizeof(int),1,fileOut);
}

//helper function to read grid from a binary file
void readDoublesBinary(FILE *fileIn,double *array,int arrayLen){
    fread(array,sizeof(double),arrayLen,fileIn);
}

//helper function to write grid to a binary file
void writeDoublesBinary(FILE *fileOut,double *array,int arrayLen){
    fwrite(array,sizeof(double),arrayLen,fileOut);
}



//helper function to read grid from a text file
void readTextRepr(const std::string& filename,double *array){
    std::ifstream file(filename);
    std::string str;
    int index=0;

    while(std::getline(file,str)){
        if(str!="---"){
            for(int i=0;i<str.length();i++){
                //stop reading if file is greater than arrayLength
                if(index<gridArea){
                    if(str[i]!='\n'){
                        if(str[i]=='#'){
                            array[index]=1;
                        }else{
                            array[index]=0;
                        }
                        index++;
                    }
                }
            }
        }
    }
    //fill in excess space with falses if file is too short
    if(index<gridArea){
        for(int i=index;i<gridArea;i++){
            array[index]=false;
        }
    }
}

//helper function to write grid to a text file
void writeTextRepr(const std::string& filename,double *array){
    std::ofstream file(filename);
    for(int i=0;i<gridArea;i++){
        if(array[i]>0){
            file<<'#';
        }else{
            file<<' ';
        }
        if((i+1)%gridWidth==0){
            file<<'\n';
        }
        if((i+1)%(gridWidth*gridHeight)==0){
            file<<"---\n";
        }
    }
}



//check for and print cuda errors
void checkCudaError(){
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::cout << hipGetErrorString(error) << std::endl;
    }
}



int main(int argc, const char * argv[]){
    //start clock
    auto startTime = std::chrono::high_resolution_clock::now();

    //default input and output files
    std::string inFolder = "input";
    std::string outFolder = "output";

    //default simulation time
    timeSteps = 1;

    /*
    Speed testing with varying grid and block sizes, using both 3D and 2D kernel implementations
        16x16x16 grid, 2mil steps:
            3D Kernel
            16x16x1 = 18921ms
            8x8x16  = 21150ms
            8x8x8   = 19432ms
            4x4x4   = 21614ms
            2x2x2   = 24362ms

            2D Kernel
            16x16   = 19389ms
            8x8     = 19425ms
            4x4     = 19992ms

        64x64x64 grid, 2mil steps:
            3D Kernel
            16x16x1 = 44162ms
            8x8x8   = 61221ms
            8x8x4   = 62442ms
            
            2D Kernel
            16x16   = 47543ms

        2048x2048x128 grid, 100 steps:
            3D Kernel
            16x16x1 = 15065ms
            8x8x8   = 18386ms

    The fastest block size across both small and large grids appears to be 16x16x1, using the 3D kernel
    */
    blockWidth = 16;
    blockHeight = 16;
    blockDepth = 1;

    //default grid configuration
    gridWidth = 16;
    gridHeight = 16;
    gridDepth = 16;



    //handle command line arguments to modify default configuration
    int optionLen = 0;
    for(int i=1;i<argc;i+=optionLen){
        if(strcmp(argv[i],"-i")==0){
            optionLen = 2;
            if(i+optionLen<=argc){
                inFolder = argv[i+1];
            }else{
                printf("Error: Missing arguments for -i\n");
                return 1;
            }
        }else if(strcmp(argv[i],"-o")==0){
            optionLen = 2;
            if(i+optionLen<=argc){
                outFolder = argv[i+1];
            }else{
                printf("Error: Missing arguments for -o\n");
                return 1;
            }
        }else if(strcmp(argv[i],"-t")==0){
            optionLen = 2;
            if(i+optionLen<=argc){
                timeSteps = strtol(argv[i+1],NULL,10);
            }else{
                printf("Error: Missing arguments for -t\n");
                return 1;
            }
        }else if(strcmp(argv[i],"-g")==0){
            optionLen = 4;
            if(i+optionLen<=argc){
                gridWidth = strtol(argv[i+1],NULL,10);
                gridHeight = strtol(argv[i+2],NULL,10);
                gridDepth = strtol(argv[i+3],NULL,10);
            }else{
                printf("Error: Missing arguments for -g\n");
                return 1;
            }
        }else if(strcmp(argv[i],"-b")==0){
            optionLen = 4;
            if(i+optionLen<=argc){
                blockWidth = strtol(argv[i+1],NULL,10);
                blockHeight = strtol(argv[i+2],NULL,10);
                blockDepth = strtol(argv[i+3],NULL,10);
            }else{
                printf("Error: Missing arguments for -b\n");
                return 1;
            }
        }else{
            printf("Error: Parameters must be of form:\n");
            printf("./game [-i infile] [-o outfile] [-t timesteps] [-g gridsize] [-b blockdimensions]\n");
            return 1;
        }
    }

    

    //print for debugging purposes
    std::cout << "In folder = " << inFolder << "\n";
    std::cout << "Out folder = " << outFolder << "\n";
    printf("Time steps = %d\n",timeSteps);
    printf("Block dimensions = %dx%dx%d\n",blockWidth,blockHeight,blockDepth);



    //read binary header
    char *inFile = (char *)calloc(inFolder.length()+strlen(SIM_STATE_NAME)+1, sizeof(char));
    strcpy(inFile,inFolder.c_str());
    strcat(inFile,SIM_STATE_NAME);

    FILE *inGridFile = fopen(inFile,"rb");
    if(inGridFile!=NULL){
        readHeaderBinary(inGridFile,&gridWidth,&gridHeight,&gridDepth);

        //print for debugging purposes
        printf("Using grid settings from file...\n");
    }else{
        //print for debugging purposes
        printf("No file found, using default grid settings...\n");
    }

    //derived values
    gridWidthBlocks = std::ceil((float)gridWidth/(float)blockWidth);
    gridHeightBlocks = std::ceil((float)gridHeight/(float)blockHeight);
    gridDepthBlocks = std::ceil((float)gridDepth/(float)blockDepth);
    gridArea = gridWidth*gridHeight*gridDepth;



    //print for debugging purposes
    printf("Grid dimensions = %dx%dx%d\n",gridWidth,gridHeight,gridDepth);
    printf("Grid in blocks = %dx%dx%d\n",gridWidthBlocks,gridHeightBlocks,gridDepthBlocks);

    

    //set device symbols to dimensions of grid,block,etc.
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridWidth_d)),&gridWidth,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridHeight_d)),&gridHeight,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridDepth_d)),&gridDepth,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&blockWidth_d)),&blockWidth,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&blockHeight_d)),&blockHeight,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&blockDepth_d)),&blockDepth,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridWidthBlocks_d)),&gridWidthBlocks,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridHeightBlocks_d)),&gridHeightBlocks,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridDepthBlocks_d)),&gridDepthBlocks,sizeof(int),0,hipMemcpyHostToDevice);

    dim3 numBlocks(gridWidthBlocks,gridHeightBlocks,gridDepthBlocks);
    dim3 blockSize(blockWidth,blockHeight,blockDepth);

    size_t gridSize = gridWidth*gridHeight*gridDepth;

    //device+host grid arrays
    double *grid_h,*grid1_h;
    double *grid_d,*grid1_d;

    //allocate host memory
    grid_h = (double *)calloc(gridSize,sizeof(double));
    grid1_h = (double *)calloc(gridSize,sizeof(double));

    //grid1_h is initialized with all zeros but in the future it may need to be set
    //(the n-2th time value it stores is used in calculation)

    //load grid_h from file
    if(inGridFile!=NULL){
        readDoublesBinary(inGridFile,grid_h,gridArea);
        fclose(inGridFile);

        //print for debugging purposes
        printf("Read grid from file...\n");
    }else{
        //print for debugging purposes
        printf("No file found, using an empty grid...\n");
    }

    //allocate device memory
    hipMalloc((void **)&grid_d, gridSize);
    hipMalloc((void **)&grid1_d, gridSize);
    
    //copy both grids to device
    hipMemcpy(grid_d,grid_h,gridSize,hipMemcpyHostToDevice);
    hipMemcpy(grid1_d,grid1_h,gridSize,hipMemcpyHostToDevice);

    for(int i=0;i<timeSteps;i++){
        solver<<<numBlocks,blockSize>>>(grid_d,grid1_d);
        hipDeviceSynchronize();
        std::swap(grid_d,grid1_d);

        checkCudaError();
    }

    //only copy first grid to host, since it was computed and then swapped by kernel
    hipMemcpy(grid_h,grid_d,gridSize,hipMemcpyDeviceToHost);



    //output grid in text form for debugging
    writeTextRepr(outFolder+"/text_repr.txt",grid_h);

    
    
    //write output binary file
    char *outFile = (char *)calloc(outFolder.length()+strlen(SIM_STATE_NAME)+1, sizeof(char));
    strcpy(outFile,outFolder.c_str());
    strcat(outFile,SIM_STATE_NAME);

    FILE *outGridFile = fopen(outFile,"wb");
    if(outGridFile!=NULL){
        writeHeaderBinary(outGridFile,&gridWidth,&gridHeight,&gridDepth);
        writeDoublesBinary(outGridFile,grid_h,gridArea);
        fclose(outGridFile);
    }

    //free host memory
    free(grid_h);
    free(grid1_h);
    //free device memory
    hipFree(grid_d);
    hipFree(grid1_d);

    //end clock
    auto endTime = std::chrono::high_resolution_clock::now();
    auto timePassed = std::chrono::duration_cast<std::chrono::milliseconds>(endTime-startTime).count();
    printf("Ran in %ld ms\n",timePassed);

    return 0;
}