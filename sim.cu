
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
#include<cmath>
#include<fstream>
#include<chrono>



//shared host/device constants
int gridWidth,gridHeight,gridDepth,blockWidth,blockHeight,blockDepth,gridWidthBlocks,gridHeightBlocks,gridDepthBlocks,gridArea;
__constant__ int gridWidth_d,gridHeight_d,gridDepth_d,blockWidth_d,blockHeight_d,blockDepth_d,gridWidthBlocks_d,gridHeightBlocks_d,gridDepthBlocks_d;

//host only constants
int timeSteps;



__device__ int arrayPos(const int &x,const int &y,const int &z){
    return (z*gridWidth_d*gridHeight_d)+(y*gridWidth_d)+x;
}



__global__ void solver(double *grid,double *grid1){
    int xpos = (blockIdx.x*blockWidth_d)+threadIdx.x;
    int ypos = (blockIdx.y*blockHeight_d)+threadIdx.y;
    int zpos = (blockIdx.z*blockDepth_d)+threadIdx.z;

    if(xpos>0 && xpos<gridWidth_d-1 && ypos>0 && ypos<gridHeight_d-1 && zpos>0 && zpos<gridDepth_d-1){
        //the grid1 value at each point contains the value from 2 steps ago(but is overwritten once used)
        grid1[arrayPos(xpos,ypos,zpos)] = grid[arrayPos(xpos+1,ypos,zpos)]+grid[arrayPos(xpos-1,ypos,zpos)]
                                        +grid[arrayPos(xpos,ypos+1,zpos)]+grid[arrayPos(xpos,ypos-1,zpos)]
                                        +grid[arrayPos(xpos,ypos,zpos+1)]+grid[arrayPos(xpos,ypos,zpos-1)]
                                        -grid1[arrayPos(xpos,ypos,zpos)];
    }
}



//helper function to read header from a binary file
void readHeaderBinary(std::ifstream &fileIn,int &w,int &h,int &d){
    fileIn.read(reinterpret_cast<char*>(&w),sizeof(int));
    fileIn.read(reinterpret_cast<char*>(&h),sizeof(int));
    fileIn.read(reinterpret_cast<char*>(&d),sizeof(int));
}

//helper function to write header to a binary file
void writeHeaderBinary(std::ofstream &fileOut,int w,int h,int d){
    fileOut.write(reinterpret_cast<char*>(&w),sizeof(int));
    fileOut.write(reinterpret_cast<char*>(&h),sizeof(int));
    fileOut.write(reinterpret_cast<char*>(&d),sizeof(int));
}

//helper function to read grid from a binary file
void readDoublesBinary(std::ifstream &fileIn,double *array,int arrayLen){
    for(int i=0;i<arrayLen;i++){
        fileIn.read(reinterpret_cast<char*>(&array[i]),sizeof(double));
    }
}

//helper function to write grid to a binary file
void writeDoublesBinary(std::ofstream &fileOut,double *array,int arrayLen){
    for(int i=0;i<arrayLen;i++){
        fileOut.write(reinterpret_cast<char*>(&array[i]),sizeof(double));
    }
}



//helper function to read grid from a text file
void readTextRepr(const std::string& filename,double *array){
    std::ifstream file(filename);
    std::string str;
    int index=0;

    while(std::getline(file,str)){
        if(str!="---"){
            for(int i=0;i<str.length();i++){
                //stop reading if file is greater than arrayLength
                if(index<gridArea){
                    if(str[i]!='\n'){
                        if(str[i]=='#'){
                            array[index]=1;
                        }else{
                            array[index]=0;
                        }
                        index++;
                    }
                }
            }
        }
    }
    //fill in excess space with falses if file is too short
    if(index<gridArea){
        for(int i=index;i<gridArea;i++){
            array[index]=false;
        }
    }
}



//helper function to write grid to a text file
void writeTextRepr(const std::string& filename,double *array){
    std::ofstream file(filename);
    for(int i=0;i<gridArea;i++){
        if(array[i]>0){
            file<<'#';
        }else{
            file<<' ';
        }
        if((i+1)%gridWidth==0){
            file<<'\n';
        }
        if((i+1)%(gridWidth*gridHeight)==0){
            file<<"---\n";
        }
    }
}



//check for and print cuda errors
void checkCudaError(){
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::cout << hipGetErrorString(error) << std::endl;
    }
}



int main(int argc, const char * argv[]){
    //start clock
    auto startTime = std::chrono::high_resolution_clock::now();

    //default input and output files
    std::string inFolder = "input";
    std::string outFolder = "output";

    //default simulation time
    timeSteps = 1;

    /*
    Speed testing with varying grid and block sizes, using both 3D and 2D kernel implementations
        16x16x16 grid, 2mil steps:
            3D Kernel
            16x16x1 = 18921ms
            8x8x16  = 21150ms
            8x8x8   = 19432ms
            4x4x4   = 21614ms
            2x2x2   = 24362ms

            2D Kernel
            16x16   = 19389ms
            8x8     = 19425ms
            4x4     = 19992ms

        64x64x64 grid, 2mil steps:
            3D Kernel
            16x16x1 = 44162ms
            8x8x8   = 61221ms
            8x8x4   = 62442ms
            
            2D Kernel
            16x16   = 47543ms

        2048x2048x128 grid, 100 steps:
            3D Kernel
            16x16x1 = 15065ms
            8x8x8   = 18386ms

    The fastest block size across both small and large grids appears to be 16x16x1, using the 3D kernel
    */
    blockWidth = 16;
    blockHeight = 16;
    blockDepth = 1;



    //handle command line arguments to modify default configuration
    int optionLen = 0;
    for(int i=1;i<argc;i+=optionLen){
        if(strcmp(argv[i],"-i")==0){
            optionLen = 2;
            if(i+optionLen<=argc){
                inFolder = argv[i+1];
            }else{
                printf("Error: Missing arguments for -i\n");
                return 1;
            }
        }else if(strcmp(argv[i],"-o")==0){
            optionLen = 2;
            if(i+optionLen<=argc){
                outFolder = argv[i+1];
            }else{
                printf("Error: Missing arguments for -o\n");
                return 1;
            }
        }else if(strcmp(argv[i],"-t")==0){
            optionLen = 2;
            if(i+optionLen<=argc){
                timeSteps = strtol(argv[i+1],NULL,10);
            }else{
                printf("Error: Missing arguments for -t\n");
                return 1;
            }
        }else if(strcmp(argv[i],"-b")==0){
            optionLen = 4;
            if(i+optionLen<=argc){
                blockWidth = strtol(argv[i+1],NULL,10);
                blockHeight = strtol(argv[i+2],NULL,10);
                blockDepth = strtol(argv[i+3],NULL,10);
            }else{
                printf("Error: Missing arguments for -b\n");
                return 1;
            }
        }else{
            printf("Error: Parameters must be of form:\n");
            printf("./game [-i infile] [-o outfile] [-t timesteps] [-b blockdimensions]\n");
            return 1;
        }
    }



    //default grid configuration
    gridWidth = 16;
    gridHeight = 16;
    gridDepth = 16;

    //read binary header
    std::ifstream inGridFile(inFolder+"/sim_state.bin",std::ofstream::binary);
    if(inGridFile.good()){
        readHeaderBinary(inGridFile,gridWidth,gridHeight,gridDepth);
        printf("TEST: %d %d %d",gridWidth,gridHeight,gridDepth);
    }

    //derived values
    gridWidthBlocks = std::ceil((float)gridWidth/(float)blockWidth);
    gridHeightBlocks = std::ceil((float)gridHeight/(float)blockHeight);
    gridDepthBlocks = std::ceil((float)gridDepth/(float)blockDepth);
    gridArea = gridWidth*gridHeight*gridDepth;



    //print everything for debugging purposes
    std::cout << "In folder = " << inFolder << "\n";
    std::cout << "Out folder = " << outFolder << "\n";
    printf("Time steps = %d\n",timeSteps);
    printf("Grid dimensions = %dx%dx%d\n",gridWidth,gridHeight,gridDepth);
    printf("Block dimensions = %dx%dx%d\n",blockWidth,blockHeight,blockDepth);
    printf("Grid in blocks = %dx%dx%d\n",gridWidthBlocks,gridHeightBlocks,gridDepthBlocks);
    printf("...\n");

    

    //set device symbols to dimensions of grid,block,etc.
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridWidth_d)),&gridWidth,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridHeight_d)),&gridHeight,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridDepth_d)),&gridDepth,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&blockWidth_d)),&blockWidth,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&blockHeight_d)),&blockHeight,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&blockDepth_d)),&blockDepth,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridWidthBlocks_d)),&gridWidthBlocks,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridHeightBlocks_d)),&gridHeightBlocks,sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(*(&gridDepthBlocks_d)),&gridDepthBlocks,sizeof(int),0,hipMemcpyHostToDevice);

    dim3 numBlocks(gridWidthBlocks,gridHeightBlocks,gridDepthBlocks);
    dim3 blockSize(blockWidth,blockHeight,blockDepth);

    size_t gridSize = gridWidth*gridHeight*gridDepth;

    //device+host grid arrays
    double *grid_h,*grid1_h;
    double *grid_d,*grid1_d;

    //allocate host memory
    grid_h = (double *)calloc(gridSize,sizeof(double));
    grid1_h = (double *)calloc(gridSize,sizeof(double));

    //grid1_h is initialized with all zeros but in the future it may need to be set
    //(the n-2th time value it stores is used in calculation)

    //load grid_h from file
    if(inGridFile.good()){
        readDoublesBinary(inGridFile,grid_h,gridArea);
        inGridFile.close();
    }

    //allocate device memory
    hipMalloc((void **)&grid_d, gridSize);
    hipMalloc((void **)&grid1_d, gridSize);

    checkCudaError();
    
    //copy both grids to device
    hipMemcpy(grid_d,grid_h,gridSize,hipMemcpyHostToDevice);
    hipMemcpy(grid1_d,grid1_h,gridSize,hipMemcpyHostToDevice);

    checkCudaError();

    for(int i=0;i<timeSteps;i++){
        solver<<<numBlocks,blockSize>>>(grid_d,grid1_d);
        hipDeviceSynchronize();
        std::swap(grid_d,grid1_d);

        checkCudaError();
    }

    //only copy first grid to host, since it was computed and then swapped by kernel
    hipMemcpy(grid_h,grid_d,gridSize,hipMemcpyDeviceToHost);

    checkCudaError();

    //output grid in text form for debugging
    //
    //
    //
    //
    //
    //
    //
    writeTextRepr(outFolder+"/text_repr.txt",grid_h);

    //write output binary file
    std::ofstream outGridFile(outFolder+"/sim_state.bin",std::ofstream::binary);
    if(outGridFile.good()){
        writeHeaderBinary(outGridFile,gridWidth,gridHeight,gridDepth);
        writeDoublesBinary(outGridFile,grid_h,gridArea);
        outGridFile.close();
    }

    //free host memory
    free(grid_h);
    //free device memory
    hipFree(grid_d);
    hipFree(grid1_d);

    checkCudaError();

    //end clock
    auto endTime = std::chrono::high_resolution_clock::now();
    auto timePassed = std::chrono::duration_cast<std::chrono::milliseconds>(endTime-startTime).count();
    printf("Ran in %ld ms\n",timePassed);

    return 0;
}